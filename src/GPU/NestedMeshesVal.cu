#include "hip/hip_runtime.h"
#ifndef _NESTED_MESHES_VAL_H_
#define _NESTED_MESHES_VAL_H_

#include "NestedMeshesCommon.cu"

/**************************************/
/* Interpolaci�n con valores directos */
/**************************************/

__device__ void obtenerDatosEInterpolarEspesoresVal(double2 *datosVolumenesNivel0_1, double2 *datosVolumenesNivel0Sig_1,
				int numVolxNivel0, int xi, int yi, double peso1, double2 *res1)
{
	double2 datos1, datos_sig1;
	int pos;
	double h, eta;
	// peso1: peso del estado actual, peso2: peso del siguiente estado
	double peso2 = 1.0-peso1;

	pos = (yi + 2)*(numVolxNivel0 + 4) + (xi + 2);
	datos1 = datosVolumenesNivel0_1[pos];
	datos_sig1 = datosVolumenesNivel0Sig_1[pos];
	h = peso1*datos1.x + peso2*datos_sig1.x;
	eta = peso1*(datos1.x-datos1.y) + peso2*(datos_sig1.x-datos_sig1.y);

	res1->x = ((h < EPSILON) ? 0.0 : h);
	res1->y = res1->x - eta;  // H = h-eta
}

__device__ void obtenerDatosEInterpolarCaudalesVal(double2 *datosVolumenesNivel0_2, double2 *datosVolumenesNivel0Sig_2,
				double2 datosNivel1_1, int numVolxNivel0, int xi, int yi, double peso1, double2 *res2)
{
	double2 datos2, datos_sig2;
	int pos;
	// peso1: peso del estado actual, peso2: peso del siguiente estado
	double peso2 = 1.0-peso1;

	pos = (yi + 2)*(numVolxNivel0 + 4) + (xi + 2);
	datos2 = datosVolumenesNivel0_2[pos];
	datos_sig2 = datosVolumenesNivel0Sig_2[pos];

	if (datosNivel1_1.x == 0.0) {
		// No hay agua
		res2->x = res2->y = 0.0;
	}
	else {
		// Hay agua
		res2->x = peso1*datos2.x + peso2*datos_sig2.x;
		res2->y = peso1*datos2.y + peso2*datos_sig2.y;
	}
}

__global__ void obtenerSiguientesEspesoresVolumenesFantasmaValoresNivel1GPU(double2 *datosVolumenesNivel0_1,
				double2 *datosVolumenesNivel0Sig_1, double2 *datosVolumenesNivel1_1, int numVolxTotalNivel0,
				int numVolyTotalNivel0, int inix, int iniy, int numVolxSubmallaNivel1, int numVolySubmallaNivel1,
				int *posCopiaNivel1, double2 *datosCopiaNivel1_1, double peso1, int ratio_ref, double borde_sup,
				double borde_inf, double borde_izq, double borde_der, int inixSubmallaSupCluster,
				int iniySubmallaSupCluster, int numVolxSubmallaSupCluster, int inixSubmallaCluster,
				int iniySubmallaCluster, bool ultima_hebraX_submalla, bool ultima_hebraY_submalla)
{
	int posNivel1;
	int pos_ghost, pos2;
	int xi, yi;
	int finx, finy;
	double2 res1;
	int pos_x_hebra, pos_y_hebra;
	bool procesada = false;
	// peso1 es el peso del estado actual (no del siguiente estado)

	pos_x_hebra = blockIdx.x*NUM_HEBRAS_ANCHO_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRAS_ALTO_EST + threadIdx.y;

	// datosVolumenesNivel0 contiene los datos del estado actual del nivel 0, mientras que
	// datosVolumenesNivel0Sig contiene los datos del siguiente estado del nivel 0 (para
	// poder interpolar en tiempo). datosVolumenesNivel1 s�lo contiene los datos de la
	// submalla actual. datosCopiaNivel1 contiene los datos de todas las submallas del
	// nivel 1, y posCopiaNivel1 indica la posici�n de datosCopiaNivel1 desde donde hay que
	// copiar los datos del volumen fastasma que trate la hebra (-1 si hay que interpolar).

	if ((pos_y_hebra > 1) && (pos_y_hebra < numVolySubmallaNivel1+2)) {
		posNivel1 = pos_y_hebra*(numVolxSubmallaNivel1+4) + pos_x_hebra;
		finx = inix + inixSubmallaCluster + numVolxSubmallaNivel1 - 1;
		if ((pos_x_hebra < 2) && (inixSubmallaCluster == 0)) {
			// Celda fantasma izquierda (de las dos primeras columnas)
			procesada = true;
			pos_ghost = inix - 1;
			if (pos_ghost > -1) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res1 = datosCopiaNivel1_1[pos2];
				}
				else {
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = iniy + iniySubmallaCluster + pos_y_hebra-2;  // (se le resta 2 por las 2 primeras celdas fantasma en y)
					xi = pos_ghost/ratio_ref - inixSubmallaSupCluster;
					yi = pos2/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarEspesoresVal(datosVolumenesNivel0_1, datosVolumenesNivel0Sig_1,
						numVolxSubmallaSupCluster, xi, yi, peso1, &res1);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = ((pos_x_hebra == 0) ? 2 : 1);
				res1 = datosVolumenesNivel1_1[posNivel1 + pos2];
			}
			datosVolumenesNivel1_1[posNivel1] = res1;
		}
		else if (((pos_x_hebra == numVolxSubmallaNivel1+2) || (pos_x_hebra == numVolxSubmallaNivel1+3)) && ultima_hebraX_submalla) {
			// Celda fantasma derecha (de las dos �ltimas columnas)
			procesada = true;
			pos_ghost = finx + 1;
			if (pos_ghost < numVolxTotalNivel0*ratio_ref) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res1 = datosCopiaNivel1_1[pos2];
				}
				else {
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = iniy + iniySubmallaCluster + pos_y_hebra-2;
					xi = pos_ghost/ratio_ref - inixSubmallaSupCluster;
					yi = pos2/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarEspesoresVal(datosVolumenesNivel0_1, datosVolumenesNivel0Sig_1,
						numVolxSubmallaSupCluster, xi, yi, peso1, &res1);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = pos_x_hebra - numVolxSubmallaNivel1 - 1;
				res1 = datosVolumenesNivel1_1[posNivel1 - pos2];
			}
			datosVolumenesNivel1_1[posNivel1] = res1;
		}
	}
	if ((! procesada) && (pos_x_hebra > 1) && (pos_x_hebra < numVolxSubmallaNivel1+2)) {
		posNivel1 = pos_y_hebra*(numVolxSubmallaNivel1+4) + pos_x_hebra;
		finy = iniy + iniySubmallaCluster + numVolySubmallaNivel1 - 1;
		if ((pos_y_hebra < 2) && (iniySubmallaCluster == 0)) {
			// Celda fantasma superior (de las dos primeras filas)
			pos_ghost = iniy - 1;
			if (pos_ghost > -1) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res1 = datosCopiaNivel1_1[pos2];
				}
				else {
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = inix + inixSubmallaCluster + pos_x_hebra-2;
					xi = pos2/ratio_ref - inixSubmallaSupCluster;
					yi = pos_ghost/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarEspesoresVal(datosVolumenesNivel0_1, datosVolumenesNivel0Sig_1,
						numVolxSubmallaSupCluster, xi, yi, peso1, &res1);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = ((pos_y_hebra == 0) ? 2 : 1);
				pos2 = pos2*(numVolxSubmallaNivel1+4);
				res1 = datosVolumenesNivel1_1[posNivel1 + pos2];
			}
			datosVolumenesNivel1_1[posNivel1] = res1;
		}
		else if (((pos_y_hebra == numVolySubmallaNivel1+2) || (pos_y_hebra == numVolySubmallaNivel1+3)) && ultima_hebraY_submalla) {
			// Celda fantasma inferior (de las dos �ltimas filas)
			pos_ghost = finy + 1;
			if (pos_ghost < numVolyTotalNivel0*ratio_ref) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res1 = datosCopiaNivel1_1[pos2];
				}
				else {
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = inix + inixSubmallaCluster + pos_x_hebra-2;
					xi = pos2/ratio_ref - inixSubmallaSupCluster;
					yi = pos_ghost/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarEspesoresVal(datosVolumenesNivel0_1, datosVolumenesNivel0Sig_1,
						numVolxSubmallaSupCluster, xi, yi, peso1, &res1);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = pos_y_hebra - numVolySubmallaNivel1 - 1;
				pos2 = pos2*(numVolxSubmallaNivel1+4);
				res1 = datosVolumenesNivel1_1[posNivel1 - pos2];
			}
			datosVolumenesNivel1_1[posNivel1] = res1;
		}
	}
}

__global__ void obtenerSiguientesCaudalesVolumenesFantasmaValoresNivel1GPU(double2 *datosVolumenesNivel0_2,
				double2 *datosVolumenesNivel0Sig_2, double2 *datosVolumenesNivel1_1, double2 *datosVolumenesNivel1_2,
				int numVolxTotalNivel0, int numVolyTotalNivel0, int inix, int iniy, int numVolxSubmallaNivel1,
				int numVolySubmallaNivel1, int *posCopiaNivel1, double2 *datosCopiaNivel1_2, double peso1,
				int ratio_ref, double borde_sup, double borde_inf, double borde_izq, double borde_der,
				int inixSubmallaSupCluster, int iniySubmallaSupCluster, int numVolxSubmallaSupCluster,
				int inixSubmallaCluster, int iniySubmallaCluster, bool ultima_hebraX_submalla, bool ultima_hebraY_submalla)
{
	int posNivel1;
	int pos_ghost, pos2;
	int xi, yi;
	int finx, finy;
	double2 datosNivel1_1;
	double2 res2;
	int pos_x_hebra, pos_y_hebra;
	bool procesada = false;
	// peso1 es el peso del estado actual (no del siguiente estado)

	pos_x_hebra = blockIdx.x*NUM_HEBRAS_ANCHO_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRAS_ALTO_EST + threadIdx.y;

	// datosVolumenesNivel0 contiene los datos del estado actual del nivel 0, mientras que
	// datosVolumenesNivel0Sig contiene los datos del siguiente estado del nivel 0 (para
	// poder interpolar en tiempo). datosVolumenesNivel1 s�lo contiene los datos de la
	// submalla actual. datosCopiaNivel1 contiene los datos de todas las submallas del
	// nivel 1, y posCopiaNivel1 indica la posici�n de datosCopiaNivel1 desde donde hay que
	// copiar los datos del volumen fastasma que trate la hebra (-1 si hay que interpolar).

	if ((pos_y_hebra > 1) && (pos_y_hebra < numVolySubmallaNivel1+2)) {
		posNivel1 = pos_y_hebra*(numVolxSubmallaNivel1+4) + pos_x_hebra;
		finx = inix + inixSubmallaCluster + numVolxSubmallaNivel1 - 1;
		if ((pos_x_hebra < 2) && (inixSubmallaCluster == 0)) {
			// Celda fantasma izquierda (de las dos primeras columnas)
			procesada = true;
			pos_ghost = inix - 1;
			if (pos_ghost > -1) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res2 = datosCopiaNivel1_2[pos2];
				}
				else {
					datosNivel1_1 = datosVolumenesNivel1_1[posNivel1];
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = iniy + iniySubmallaCluster + pos_y_hebra-2;  // (se le resta 2 por las 2 primeras celdas fantasma en y)
					xi = pos_ghost/ratio_ref - inixSubmallaSupCluster;
					yi = pos2/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarCaudalesVal(datosVolumenesNivel0_2, datosVolumenesNivel0Sig_2,
						datosNivel1_1, numVolxSubmallaSupCluster, xi, yi, peso1, &res2);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = ((pos_x_hebra == 0) ? 2 : 1);
				res2 = datosVolumenesNivel1_2[posNivel1 + pos2];
				res2.x *= borde_izq;
			}
			datosVolumenesNivel1_2[posNivel1] = res2;
		}
		else if (((pos_x_hebra == numVolxSubmallaNivel1+2) || (pos_x_hebra == numVolxSubmallaNivel1+3)) && ultima_hebraX_submalla) {
			// Celda fantasma derecha (de las dos �ltimas columnas)
			procesada = true;
			pos_ghost = finx + 1;
			if (pos_ghost < numVolxTotalNivel0*ratio_ref) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res2 = datosCopiaNivel1_2[pos2];
				}
				else {
					datosNivel1_1 = datosVolumenesNivel1_1[posNivel1];
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = iniy + iniySubmallaCluster + pos_y_hebra-2;
					xi = pos_ghost/ratio_ref - inixSubmallaSupCluster;
					yi = pos2/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarCaudalesVal(datosVolumenesNivel0_2, datosVolumenesNivel0Sig_2,
						datosNivel1_1, numVolxSubmallaSupCluster, xi, yi, peso1, &res2);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = pos_x_hebra - numVolxSubmallaNivel1 - 1;
				res2 = datosVolumenesNivel1_2[posNivel1 - pos2];
				res2.x *= borde_der;
			}
			datosVolumenesNivel1_2[posNivel1] = res2;
		}
	}
	if ((! procesada) && (pos_x_hebra > 1) && (pos_x_hebra < numVolxSubmallaNivel1+2)) {
		posNivel1 = pos_y_hebra*(numVolxSubmallaNivel1+4) + pos_x_hebra;
		finy = iniy + iniySubmallaCluster + numVolySubmallaNivel1 - 1;
		if ((pos_y_hebra < 2) && (iniySubmallaCluster == 0)) {
			// Celda fantasma superior (de las dos primeras filas)
			pos_ghost = iniy - 1;
			if (pos_ghost > -1) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res2 = datosCopiaNivel1_2[pos2];
				}
				else {
					datosNivel1_1 = datosVolumenesNivel1_1[posNivel1];
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = inix + inixSubmallaCluster + pos_x_hebra-2;
					xi = pos2/ratio_ref - inixSubmallaSupCluster;
					yi = pos_ghost/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarCaudalesVal(datosVolumenesNivel0_2, datosVolumenesNivel0Sig_2,
						datosNivel1_1, numVolxSubmallaSupCluster, xi, yi, peso1, &res2);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = ((pos_y_hebra == 0) ? 2 : 1);
				pos2 = pos2*(numVolxSubmallaNivel1+4);
				res2 = datosVolumenesNivel1_2[posNivel1 + pos2];
				res2.y *= borde_sup;
			}
			datosVolumenesNivel1_2[posNivel1] = res2;
		}
		else if (((pos_y_hebra == numVolySubmallaNivel1+2) || (pos_y_hebra == numVolySubmallaNivel1+3)) && ultima_hebraY_submalla) {
			// Celda fantasma inferior (de las dos �ltimas filas)
			pos_ghost = finy + 1;
			if (pos_ghost < numVolyTotalNivel0*ratio_ref) {
				pos2 = posCopiaNivel1[posNivel1];
				if (pos2 > -1) {
					res2 = datosCopiaNivel1_2[pos2];
				}
				else {
					datosNivel1_1 = datosVolumenesNivel1_1[posNivel1];
					// (xi, yi): Posici�n de la celda asociada del nivel 0
					pos2 = inix + inixSubmallaCluster + pos_x_hebra-2;
					xi = pos2/ratio_ref - inixSubmallaSupCluster;
					yi = pos_ghost/ratio_ref - iniySubmallaSupCluster;
					obtenerDatosEInterpolarCaudalesVal(datosVolumenesNivel0_2, datosVolumenesNivel0Sig_2,
						datosNivel1_1, numVolxSubmallaSupCluster, xi, yi, peso1, &res2);
				}
			}
			else {
				// Celda que se sale del dominio
				pos2 = pos_y_hebra - numVolySubmallaNivel1 - 1;
				pos2 = pos2*(numVolxSubmallaNivel1+4);
				res2 = datosVolumenesNivel1_2[posNivel1 - pos2];
				res2.y *= borde_inf;
			}
			datosVolumenesNivel1_2[posNivel1] = res2;
		}
	}
}

/*************************/
/* Proyecci�n con medias */
/*************************/

__global__ void corregirSolucionSubmallaValoresNivel1GPU(double2 *d_datosVolumenesNivel0Sig_1, double2 *d_datosVolumenesNivel0Sig_2,
				double2 *d_datosVolumenesNivel1_1, double2 *d_datosVolumenesNivel1_2, int inix, int iniy, int numVolxNivel1,
				int numVolyNivel1, int numVolxNivel0, int ratio_ref, double factor_correccion, int inixSubmallaSupCluster,
				int iniySubmallaSupCluster, int inixSubmallaCluster, int iniySubmallaCluster)
{
	int i, j, k, l;
	int xi, yi;
	int posNivel0, posNivel1;
	double2 datosNivel0_1, media_q;
	double h, media_h, media_eta;
	int pos_x_hebra, pos_y_hebra;
	int finsx, finsy;
	bool hay_celda_seca = false;

	__shared__ double2 s_datosNivel1_1[NUM_HEBRAS_ANCHO_EST][NUM_HEBRAS_ALTO_EST];
	__shared__ double2 s_datosNivel1_2[NUM_HEBRAS_ANCHO_EST][NUM_HEBRAS_ALTO_EST];

	pos_x_hebra = blockIdx.x*NUM_HEBRAS_ANCHO_EST + threadIdx.x;
	pos_y_hebra = blockIdx.y*NUM_HEBRAS_ALTO_EST + threadIdx.y;

	// Comprobamos si la hebra est� dentro de los l�mites de la malla
	if ((pos_x_hebra < numVolxNivel1) && (pos_y_hebra < numVolyNivel1)) {
		posNivel1 = (pos_y_hebra+2)*(numVolxNivel1+4) + pos_x_hebra+2;
		i = pos_x_hebra&(NUM_HEBRAS_ANCHO_EST-1);  // pos_x_hebra%NUM_HEBRAS_ANCHO_EST
		j = pos_y_hebra&(NUM_HEBRAS_ALTO_EST-1);   // pos_y_hebra%NUM_HEBRAS_ALTO_EST
		s_datosNivel1_1[i][j] = d_datosVolumenesNivel1_1[posNivel1];
		s_datosNivel1_2[i][j] = d_datosVolumenesNivel1_2[posNivel1];
	}
	__syncthreads();

	if ((pos_x_hebra < numVolxNivel1) && (pos_y_hebra < numVolyNivel1)) {
		if (((i&(ratio_ref-1)) == 0) && ((j&(ratio_ref-1)) == 0)) {
			// La hebra correspondiente a la esquina superior izquierda de la celda gruesa
			// asociada obtiene la media de las celdas finas
			media_h = 0.0;
			media_eta = 0.0;
			media_q.x = media_q.y = 0.0;
			finsx = i + ratio_ref;
			finsy = j + ratio_ref;
			for (l=i; l<finsx; l++) {
				for (k=j; k<finsy; k++) {
					h = s_datosNivel1_1[l][k].x;
					if (h < EPSILON)
						hay_celda_seca = true;
					media_h += h;
					media_eta += h - s_datosNivel1_1[l][k].y;
					media_q.x += s_datosNivel1_2[l][k].x;
					media_q.y += s_datosNivel1_2[l][k].y;
				}
			}
			media_h *= factor_correccion;
			media_eta *= factor_correccion;
			media_q.x *= factor_correccion;
			media_q.y *= factor_correccion;

			// Actualizamos el estado de la celda gruesa del nivel 0 si todas las celdas finas estan mojadas
			if (! hay_celda_seca) {
				// Todas las celdas finas est�n mojadas
				xi = (inix + inixSubmallaCluster + pos_x_hebra)/ratio_ref - inixSubmallaSupCluster;
				yi = (iniy + iniySubmallaCluster + pos_y_hebra)/ratio_ref - iniySubmallaSupCluster;
				posNivel0 = (yi + 2)*(numVolxNivel0 + 4) + (xi + 2);
				datosNivel0_1.x = media_h;
				datosNivel0_1.y = media_h - media_eta;  // H = h-eta
				if (datosNivel0_1.x < EPSILON) {
					datosNivel0_1.x = media_q.x = media_q.y = 0.0;
				}
				d_datosVolumenesNivel0Sig_1[posNivel0] = datosNivel0_1;
				d_datosVolumenesNivel0Sig_2[posNivel0] = media_q;
			}
		}
	}
}

void corregirSolucionValoresNivelGPU(int nivel, TDatosClusterCPU datosClusterCPU[MAX_LEVELS][MAX_GRIDS_LEVEL],
				double2 *d_datosVolumenesNivel0Sig_1, double2 *d_datosVolumenesNivel0Sig_2, double *d_correccionEtaNivel0,
				double2 *d_correccionNivel0_2, double2 *d_datosVolumenesNivel1_1, double2 *d_datosVolumenesNivel1_2,
				int ratioRefNivelInf, double factorCorreccionNivelInf, double epsilon_h, int4 submallasNivel[MAX_LEVELS][MAX_GRIDS_LEVEL],
				int *numSubmallasNivel, int submallaNivelSuperior[MAX_LEVELS][MAX_GRIDS_LEVEL],
				dim3 blockGridEstNivel[MAX_LEVELS][MAX_GRIDS_LEVEL], dim3 threadBlockEst, hipStream_t *streams, int nstreams)
{
	int i, j;
	int posNivel, posNivelInf;
	int nvxNivel, nvyNivel;
	int nvxNivelInf, nvyNivelInf;
	int nivelInf = nivel+1;

	// nivel es el nivel en el que se aplica la correcci�n
	posNivel = 0;
	for (i=0; i<numSubmallasNivel[nivel]; i++) {
		if (datosClusterCPU[nivel][i].iniy != -1) {
			nvxNivel = datosClusterCPU[nivel][i].numVolx;
			nvyNivel = datosClusterCPU[nivel][i].numVoly;
			// Medias para cada celda del nivel
			posNivelInf = 0;
			for (j=0; j<numSubmallasNivel[nivelInf]; j++) {
				if (datosClusterCPU[nivelInf][j].iniy != -1) {
					nvxNivelInf = datosClusterCPU[nivelInf][j].numVolx;
					nvyNivelInf = datosClusterCPU[nivelInf][j].numVoly;
					if (submallaNivelSuperior[nivelInf][j] == i) {
						// La submalla j de NivelInf est� contenida en la submalla i del nivel
						corregirSolucionSubmallaValoresNivel1GPU<<<blockGridEstNivel[nivelInf][j], threadBlockEst, 0, streams[j&nstreams]>>>(
							d_datosVolumenesNivel0Sig_1+posNivel, d_datosVolumenesNivel0Sig_2+posNivel, d_datosVolumenesNivel1_1+posNivelInf,
							d_datosVolumenesNivel1_2+posNivelInf, submallasNivel[nivelInf][j].x, submallasNivel[nivelInf][j].y, nvxNivelInf,
							nvyNivelInf, nvxNivel, ratioRefNivelInf, factorCorreccionNivelInf, datosClusterCPU[nivel][i].inix,
							datosClusterCPU[nivel][i].iniy, datosClusterCPU[nivelInf][j].inix, datosClusterCPU[nivelInf][j].iniy);
					}
					posNivelInf += (nvxNivelInf + 4)*(nvyNivelInf + 4);
				}
			}

			// Sincronizaci�n necesaria para evitar que corrija flujos sin haber terminado la proyecci�n de la soluci�n fina
			hipDeviceSynchronize();
			// Correcci�n debido al ajuste de flujos en las fronteras entre niveles
			corregirFlujosNivelGPU<<<blockGridEstNivel[nivel][i], threadBlockEst>>>(d_datosVolumenesNivel0Sig_1+posNivel,
				d_datosVolumenesNivel0Sig_2+posNivel, d_correccionEtaNivel0+posNivel, d_correccionNivel0_2+posNivel,
				nvxNivel, nvyNivel);

			posNivel += (nvxNivel + 4)*(nvyNivel + 4);
		}
	}
}

#endif

