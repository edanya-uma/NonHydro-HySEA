
#include <hip/hip_runtime.h>
#include <stdio.h>

// Funci�n que devuelve 0 si la tarjeta gr�fica soporta CUDA, 1 si no hay tarjeta gr�fica,
// y 2 si hay pero no soporta CUDA
extern "C" int comprobarSoporteCUDA()
{
    int valor, dev, deviceCount;

    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
        valor = 1;
	else {
		for (dev=0; dev < deviceCount; ++dev) {
	        hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, dev);
			if (deviceProp.major >= 1)
	            break;
	    }
		if (dev == deviceCount)
			valor = 2;
		else
			valor = 0;
	}
	return valor;
}
